#include "hip/hip_runtime.h"
#include "RenderObjects/render_structs.h"
#include <hip/hip_runtime.h>
#include "Materialmanager.h"
#include "RGBColors.h"

__device__ float4 get_color(float4* texels, texture_pos* positions, uint2* dimensions,
	texture_pos pos, float u, float v) 
{
	uint2 dim = dimensions[pos];
	int iv = u * (dim.x-1);
	int iu = v * (dim.y-1);

	texture_pos index = positions[pos] + iu + iv * dim.x;
	return texels[index];
}

__device__ float4 shade(ShadeRec &sr, world_struct *world) {

	float3 wo = -sr.ray.d;

	material_params material = world->materials[sr.material];
	float4 texel_color = get_color(world->texels, world->positions, world->dimensions, material.position, sr.u, sr.v);

	// ==== Simple Ambient ======
	// lambertian rho
	float4 L = scale_color(texel_color, material.ka);
	
	// ==== sun: ================
	float ndotwi = sr.normal * world->light_dir;

	if (ndotwi > 0.f) {
		L = add_colors(L, scale_color(texel_color, material.kd * invPI * world->light_intensity * ndotwi));
	}

	
	/*int numLights = sr.w->lights.size();

	for (int j = 0; j < numLights; j++) {
		Vector wi = sr.w->lights[j]->get_direction(sr);
		real ndotwi = sr.normal * wi;



		if (ndotwi > 0.0) {
			bool in_shadow = false;
			if (sr.w->lights[j]->casts_shadows())
			{
				Ray shadowray(sr.local_hit_point + kEpsilon * sr.normal, wi);
				in_shadow = sr.w->lights[j]->in_shadow(shadowray, sr);
			}

			if (!in_shadow)
				L += diffuse_brdf->f(sr, wo, wi) * sr.w->lights[j]->L(sr) * ndotwi;
		}
	}


	if (has_transparency) {
		Ray second_ray(sr.local_hit_point + kEpsilon * sr.ray.d, sr.ray.d);

		real tr = diffuse_brdf->transparency(sr);
		if (tr < 1.0)
			L = tr * L + ((real)1.0 - tr) * sr.w->tracer_ptr->trace_ray(second_ray, sr.depth + 1);
	}*/

	return L;
}