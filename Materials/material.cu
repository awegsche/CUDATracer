#include "hip/hip_runtime.h"
#include "RenderObjects/render_structs.h"
#include <hip/hip_runtime.h>
#include "Materialmanager.h"
#include "RGBColors.h"
#include "MinecraftWorld/MCWorld.cu"
#include "Samplers/sampler.h"

__device__ float4 get_color(float4* texels, texture_pos* positions, uint2* dimensions,
	texture_pos pos, float u, float v) 
{
	uint2 dim = dimensions[pos];
	int iv = u * (dim.x-1);
	int iu = v * (dim.y-1);

	texture_pos index = positions[pos] + iu + iv * dim.x;
	return texels[index];
}

__device__ bool shade(ShadeRec &sr, world_struct *world, const int seed, bool hitt, float4 &L, float4 &texel_color) {

	if (!hitt) return;

	float3 wo = -sr.ray.d;

	material_params material = world->materials[sr.material];
	texel_color = get_color(world->texels, world->positions, world->dimensions, material.position, sr.u, sr.v);

	if (material.transparent && texel_color.w < 1.0f)
		return false;

	// ==== Simple Ambient ======
	// lambertian rho
	L = scale_color(texel_color, material.ka * 0.2);
	float3 u, v, w;

	w = sr.normal;
	v = _normalize(w ^ make_float3(-0.0073f, 1.0f, 0.0034f));
	u = v ^ w;
	float3 sp = sample_hemisphere(world->smplr, seed);

	Ray shadowray;
	shadowray.o = sr.hitPoint + kEPSILON * sr.normal;
	shadowray.d = sp.x * u + sp.y * v + sp.z * w;
	ShadeRec dum;
	float tshadow = kHUGEVALUE;
	if (!world_hit(shadowray, tshadow, world, dum))
		L = scale_color(texel_color, material.ka * 1.2);

	
	/*int numLights = sr.w->lights.size();

	for (int j = 0; j < numLights; j++) {
		Vector wi = sr.w->lights[j]->get_direction(sr);
		real ndotwi = sr.normal * wi;



		if (ndotwi > 0.0) {
			bool in_shadow = false;
			if (sr.w->lights[j]->casts_shadows())
			{
				Ray shadowray(sr.local_hit_point + kEpsilon * sr.normal, wi);
				in_shadow = sr.w->lights[j]->in_shadow(shadowray, sr);
			}

			if (!in_shadow)
				L += diffuse_brdf->f(sr, wo, wi) * sr.w->lights[j]->L(sr) * ndotwi;
		}
	}


	if (has_transparency) {
		Ray second_ray(sr.local_hit_point + kEpsilon * sr.ray.d, sr.ray.d);

		real tr = diffuse_brdf->transparency(sr);
		if (tr < 1.0)
			L = tr * L + ((real)1.0 - tr) * sr.w->tracer_ptr->trace_ray(second_ray, sr.depth + 1);
	}*/

	return true;
	
}

__device__ void shade_shadow(world_struct *world, ShadeRec &sr, int seed, float4 &L, float4 &texel_color, bool hitt) {
	if (!hitt) return;
	
	// ==== sun: ================

	material_params material = world->materials[sr.material];


	float ndotwi = -sr.normal * world->light_dir;

	if (ndotwi > 0.f) {
		Ray shadowray;
		shadowray.o = sr.hitPoint + kEPSILON * sr.normal;
		shadowray.d = world->light_dir;
		float t = kHUGEVALUE;
		ShadeRec dummy;
		bool hit = world_hit(shadowray, t, world, dummy);

		if (!hit)
			L = add_colors(L, scale_color(texel_color, material.kd * invPI * world->light_intensity * ndotwi));
	}


}