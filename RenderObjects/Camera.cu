#include "hip/hip_runtime.h"
#ifndef CAMERA_CU
#define CAMERA_CU

#include "Camera.h"

#include "Materials/RGBColors.h"
#include "render_structs.h"

#include "MinecraftWorld/MCWorld.h"
#include "MinecraftWorld/MCWorld.cu"
#include "Materials/material.cu"
#include "Samplers/sampler.cu"

#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>

#define D_FACT 1.05f
#define MOVE_VALUE 10.0f
#define MAX_DEPTH 5

__device__ float3 trace_ray(
	Ray &ray, world_struct *world, const float3 &sp, int depth = 1) 
{
	float t = kHUGEVALUE;
	ShadeRec sr;

	float sky = clamp(ray.d.y * 2.1f, 0.0f, .8f);
	//float3 sky_color = rgbcolor(.8f - sky, .9f - sky, 1.0f - sky * 0.3);
	//float3 L = rgbcolor(.8f - sky, .9f - sky, 1.0f - sky * 0.3);
	rgbcol L;
	rgbacol texel_color;

	bool hit = world_hit(ray, t, world, sr);

	if (hit)
		L = shade(sr, world, sp, hit, texel_color);
	else
		depth = 10000;
	
	while(!hit && depth < MAX_DEPTH) // if transparent block, continue until non-transparent surface is hit 
	{
		depth++;
		ray.o = sr.hitPoint() + kEPSILON * ray.d;
		t = kHUGEVALUE;
		hit = world_hit(ray, t, world, sr);
		L += shade(sr, world, sp, hit, texel_color);
	}
	
	if(depth == 1)
		__syncthreads();
	shade_shadow(world, sr, sp, L, texel_color, hit);
	if(depth == 1)
		__syncthreads();
	/*
	if (depth < MAX_DEPTH) {
		float3 wi;
		shade_reflection(world, sr, sr.ray.d, wi, L, seed, depth + 1);
	}
	*/
	if (t > world->haze_dist)
	{
		float factor = 1.0f / (1.0f + (t - world->haze_dist) * world->haze_strength);

		for (int i = 0; i < world->haze_attenuation; i++)
			factor *= factor;
		L = add_colors(scale_color(L, factor), scale_color(rgbcolor(.8f - sky, .9f - sky, 1.0f - sky * 0.3), 1.0f - factor));
	}
	
	return L;
		
}

// The kernel to render with the Thinlens camera
__global__ void
//__launch_bounds__(1024, 16)
render_kernel(
	float3 *dst, const int hres, const int vres, const int seed, const float s,
	float3 eye, float3 u, float3 v, float3 w, float aperture, float d, world_struct* world)
{
	for(int block_X = 0; block_X < hres; block_X +=  gridDim.x * blockDim.x)
	{
		int ix = threadIdx.x + blockDim.x * blockIdx.x + block_X;
		int iy = threadIdx.y + blockIdx.y * blockDim.y;

		if (ix < hres) {
			int index = ix + iy * hres;
			Ray ray;
			float2 pp;      // Sample point on a pixel

			float3 L = rgbcolor(0, 0, 0);
			float2 sp = sample_square(world->smplr, seed);
			float3 hemisphere_sp = sample_hemisphere(world->smplr, seed);

			pp.x = s * (ix - 0.5 * hres + sp.x);
			pp.y = s * (iy - 0.5 * vres + sp.y);
			ray.o = eye;
			float3 dir = pp.x * u + pp.y * v - d * w;
			ray.d = _normalize(dir);
			dst[index] = add_colors(dst[index], trace_ray(ray, world, hemisphere_sp));
			__syncthreads();
		}
	}

}

__global__ void finish_kernel(uchar4 *dst, float3 *colors, const int hres, const int vres, const
        float exposure)
{
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int index = ix + iy * hres;

	dst[index] = _rgbcolor_to_byte(scale_color(colors[index], exposure));
}

__global__ void expose_kernel(
	float3 *colors, const int hres, const int vres, const float s,
	float3 eye, float3 u, float3 v, float3 w, float aperture, float d, world_struct* world, int seed)
{
	Ray ray;

	
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int index = ix + iy * hres;

	//float4 L = rgbcolor(0, 0, 0);
	float2 pp;
	float3 ap = sample_hemisphere(world->smplr, seed);

	pp.x = s * (ix - 0.5 * hres + ap.x);
	pp.y = s * (iy - 0.5 * vres + ap.y);

	ray.o = eye + (aperture * ap.x) * u + (aperture * ap.y) * v;
	ray.d = _normalize((pp.x - aperture * ap.x) * u + (pp.y - aperture * ap.y) * v - d * w);

	colors[index] = add_colors(colors[index], trace_ray(ray, world, ap));
}



Camera::Camera()
	:d(100.0f), zoom(.05f), aperture(.15f)
{
	up = make_float3(0.f, 1.f, 0.f);
}


Camera::~Camera()
{
}

void Camera::render(rgbcol* colors, const int width, const int height, const float time) const
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	//dim3 num_blocks = dim3(width / BLOCKDIM_X, height / BLOCKDIM_Y);
	dim3 num_blocks = dim3(4,  height / BLOCKDIM_Y);

	world->light_dir = _normalize(make_float3(world->light_dir.x, world->light_dir.y, sin(time * 1.0e-4f)));



	render_kernel << <num_blocks, threads >> > (
		colors, width, height, rand(), zoom,
		eye, u, v, w, aperture, d, world);


}
void Camera::expose(rgbcol* colors, const int width, const int height, const int sample_count) const
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 num_blocks = dim3(width / BLOCKDIM_X, height / BLOCKDIM_Y);

	expose_kernel << <num_blocks, threads >> > (
		colors, width, height, zoom,
		eye, u, v, w, aperture, d, world, sample_count);


}

void Camera::finish(uchar4 * frame, rgbcol * colors, const int w, const int h, const int sample_count) const
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 num_blocks = dim3(w / BLOCKDIM_X, h / BLOCKDIM_Y);



	finish_kernel << <num_blocks, threads >> > (frame, colors, w, h, 1.0f / sample_count);

}

void Camera::compute_uvw()
{
	w = eye - lookat;
	w = _normalize(w);
	u = up ^ w;
	u = _normalize(u);
	v = w ^ u;
}

void Camera::set_world(world_struct * w)
{
	world = w;
}

void Camera::set_eye(float x, float y, float z)
{
	eye = _make_float3(x, y, z);
}

void Camera::move_eye(float x, float y, float z)
{
	eye += _make_float3(x, y, z);
}

void Camera::move_eye_forward(float d)
{
	eye += w * (-d);
}

void Camera::set_lookat(float x, float y, float z)
{
	lookat = _make_float3(x, y, z);

}

void Camera::move_eye_left(float d)
{
	eye += u * d;
}

void Camera::move_eye_right(float d)
{
	eye += u * (-d);
}

void Camera::move_eye_backward(float d)
{
	eye += w * (d);
}

void Camera::rotate_up(float d)
{
	w += up * (-d);
	w = _normalize(w);
	u = up ^ w;
	u = _normalize(u);
	v = w ^ u;
}

void Camera::rotate_down(float d)
{
	w += up * (d);
	w = _normalize(w);
	u = up ^ w;
	u = _normalize(u);
	v = w ^ u;
}

void Camera::rotate_left(float d)
{
	w += u * d;
	w = _normalize(w);
	u = up ^ w;
	u = _normalize(u);
	v = w ^ u;
}

void Camera::rotate_right(float d)
{
	w += u * (-d);
	w = _normalize(w);
	u = up ^ w;
	u = _normalize(u);
	v = w ^ u;
}

void Camera::increase_d()
{
	d *= D_FACT;
	zoom *= D_FACT;
}

void Camera::decrease_d()
{
	d /= D_FACT;
	zoom /= D_FACT;
}

void Camera::zoom_in()
{
	zoom *= 1.2f;
}

void Camera::zoom_out()
{
	zoom /= 1.2f;
}

void Camera::increase_aperture()
{
	aperture *= 1.5f;
}


void Camera::decrease_aperture()
{
	aperture /= 1.5f;
}

#endif
